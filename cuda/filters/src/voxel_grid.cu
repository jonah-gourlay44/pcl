#include "hip/hip_runtime.h"
#ifdef _WIN32
# define NOMINMAX
# define WIN32_LEAN_AND_MEAN
# include <windows.h>
#endif

#include <pcl/pcl_exports.h>

#include "pcl/cuda/filters/voxel_grid.h"
#include "pcl/cuda/time_gpu.h"
#include <stdio.h>
#include <pcl/cuda/time_cpu.h>
#include <pcl/common/io.h>


namespace pcl
{
    namespace cuda
    {
        template <typename InputIteratorT> void
        getMinMax3D (InputIteratorT begin, InputIteratorT end, 
                     float4 &min_p, float4 &max_p)
        {   
            float3 init_min; 
            init_min.x = init_min.y = init_min.z = FLT_MAX;

            float3 init_max;
            init_max.x = init_max.y = init_max.z = -FLT_MAX;
            
            float3 min_pt = thrust::reduce(begin, end, init_min, Min3DKernel());
            min_p.x = min_pt.x; min_p.y = min_pt.y; min_p.z = min_pt.z; min_p.w = 0.0f;
            
            float3 max_pt = thrust::reduce(begin, end, init_max, Max3DKernel());
            max_p.x = max_pt.x; max_p.y = max_pt.y; max_p.z = max_pt.z; max_p.w = 0.0f;
        }

        template<template <typename> class Storage> void
        VoxelGrid<Storage>::applyFilter (const boost::shared_ptr <PointCloud> &output)
        {   
            if (input_->points.size () == 0)
            {
                output->width = output->height = 0;
                return;
            }
            size_t nr_points = input_->width * input_->height;

            output->height = 1;

            float4 min_p, max_p;
            PtIter begin = input_->points.begin ();
            PtIter end = input_->points.end ();

            //get 3D minimum and maximum of pointcloud
            getMinMax3D<typename PointIterator<Storage,const PointXYZRGB>::type> (begin, end, min_p, max_p);

            int64_t dx = static_cast<int64_t>((max_p.x - min_p.x) * inverse_leaf_size_.x)+1;
            int64_t dy = static_cast<int64_t>((max_p.y - min_p.y) * inverse_leaf_size_.y)+1;
            int64_t dz = static_cast<int64_t>((max_p.z - min_p.z) * inverse_leaf_size_.z)+1;

            if ( (dx*dy*dz) > static_cast<int64_t>(std::numeric_limits<int32_t>::max()) )
            {
                PCL_WARN("[pcl::cuda::VoxelGrid::applyFilter] Leaf size is too small for the input dataset. Integer indices would overflow.");
            }

            min_b_[0] = static_cast<int> (floor (min_p.x * inverse_leaf_size_.x));
            max_b_[0] = static_cast<int> (floor (max_p.x * inverse_leaf_size_.x));
            min_b_[1] = static_cast<int> (floor (min_p.y * inverse_leaf_size_.y));
            max_b_[1] = static_cast<int> (floor (max_p.y * inverse_leaf_size_.y));
            min_b_[2] = static_cast<int> (floor (min_p.z * inverse_leaf_size_.z));
            max_b_[2] = static_cast<int> (floor (max_p.z * inverse_leaf_size_.z));
            
            div_b_[0] = max_b_[0] - min_b_[0] + 1;
            div_b_[1] = max_b_[1] - min_b_[1] + 1;
            div_b_[2] = max_b_[2] - min_b_[2] + 1;
            div_b_[3] = 0;

            boost::shared_ptr<VectorCP> index_vector (new VectorCP);
            index_vector->resize (nr_points);

            divb_mul_[0] = 1;
            divb_mul_[1] = div_b_[0];
            divb_mul_[2] = div_b_[0] * div_b_[1], 0;
            divb_mul_[3] = 0;

            std::vector<float> pt (4, 0.0);

            LeafInsert<Storage> leaf_inserter = LeafInsert<Storage> (inverse_leaf_size_, min_b_, divb_mul_);

            VectorCPIter out = index_vector->begin();

            CountIter idxFirst(0);
            CountIter idxLast = idxFirst + nr_points;

            ZipIter first = thrust::make_zip_iterator(thrust::make_tuple(begin, idxFirst));
            ZipIter last = thrust::make_zip_iterator(thrust::make_tuple(end, idxLast));

            //insert points into the leafs
            thrust::transform(first, last, out, leaf_inserter);

            VectorCPIter idx_vec_start = index_vector->begin();
            VectorCPIter idx_vec_end = index_vector->end();
            
            //sort so that points with same voxel index are adjacent
            thrust::sort(idx_vec_start, idx_vec_end);

            CountIter outputFirst(0);
            CountIter outputLast = outputFirst + index_vector->size();

            ZipIterInt outputBegin = thrust::make_zip_iterator(thrust::make_tuple(idx_vec_start, outputFirst));
            ZipIterInt outputEnd = thrust::make_zip_iterator(thrust::make_tuple(idx_vec_end, outputLast));

            boost::shared_ptr<VectorCP> index_set (new VectorCP);
            index_set->resize (index_vector->size());

            //figure out original array index of each points
            thrust::transform(outputBegin, outputEnd, index_set->begin(), cloudIdxCp());

            //make a set of all unique voxel indices
            VectorCPIter vecI = thrust::unique(index_set->begin(), index_set->end(), IdxUnique());

            uint32_t total = vecI - index_set->begin();
            output->points.resize (total);
            output->width = total;

            boost::shared_ptr<CloudIndexContainer<Storage> > ind_container (new CloudIndexContainer<Storage>);
            ind_container->indices_ = *index_vector;

            centroidKernel<Storage> centroid_iter = centroidKernel<Storage> (ind_container, input_);

            //create and insert centroids into output cloud
            thrust::transform(index_set->begin(), vecI, output->points.begin(), centroid_iter);
        }

        template PCL_EXPORTS void getMinMax3D<typename PointIterator<Device,const PointXYZRGB>::type>
            (PointIterator<Device,const PointXYZRGB>::type begin,
             PointIterator<Device,const PointXYZRGB>::type end,
             float4 &min_p,
             float4 &max_p);
        template PCL_EXPORTS void getMinMax3D<typename PointIterator<Host,const PointXYZRGB>::type>
            (PointIterator<Host,const PointXYZRGB>::type begin,
             PointIterator<Host,const PointXYZRGB>::type end,
             float4 &min_p,
             float4 &max_p);
        template class PCL_EXPORTS VoxelGrid<Device>;
        template class PCL_EXPORTS VoxelGrid<Host>;
        template class PCL_EXPORTS CloudIndexContainer<Device>;
        template class PCL_EXPORTS CloudIndexContainer<Host>;
    }
}